#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <cuda/std/cmath>

namespace VMF_CUDA
{
    typedef unsigned long int cuSize; //<-- The GTX1060 3GB doesn't support size_t
    // ReLU
    template <typename vmfDevType> //<-- vmfDevType = VMF[Various Math Functions] Devoloper Type
    __global__ void ReLU(vmfDevType* x, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadIndexX < arraySize) x[threadIndexX] = x[threadIndexX] > vmfDevType(0) ? x[threadIndexX] : vmfDevType(0);
        __syncthreads();
    }

    template <typename vmfDevType>
    __global__ void ReLUDerivative(vmfDevType* x, vmfDevType y, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadIndexX < arraySize) x[threadIndexX] = x[threadIndexX] > vmfDevType(0) ? y : vmfDevType(0.0);
        __syncthreads();
    }

    // Leaky ReLU
    template <typename vmfDevType>
    __global__ void leakyReLU(vmfDevType* x, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadIndexX < arraySize) x[threadIndexX] = x[threadIndexX] < vmfDevType(0) ? x[threadIndexX] : x[threadIndexX] * vmfDevType(0.01);
        __syncthreads();
    }

    template <typename vmfDevType>
    __global__ void leakyReLUDerivative(vmfDevType* x, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadIndexX < arraySize) x[threadIndexX] = x[threadIndexX] < vmfDevType(0) ? x[threadIndexX] * 100 : x[threadIndexX];
        __syncthreads();
    }

    // Sigmoid
    template <typename vmfDevType>
    __global__ void sigmoid(vmfDevType* x, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadIndexX < arraySize) x[threadIndexX] = vmfDevType(1.0) / (vmfDevType(1.0) + exp(-x[threadIndexX]));
        __syncthreads();
    }

    template <typename vmfDevType>
    __global__ void sigmoidDerivative(vmfDevType* x, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadIndexX < arraySize)
        {
            const vmfDevType sigmoidValue = vmfDevType(1.0) / (vmfDevType(1.0) + exp(-x[threadIndexX]));
            x[threadIndexX] = sigmoidValue * (vmfDevType(1.0) - sigmoidValue);
        }
        __syncthreads();
    }


    // Heaviside step function
    template <typename vmfDevType>
    __global__ void heaviside(vmfDevType* x, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        if (threadIndexX < arraySize) x[threadIndexX] = (x[threadIndexX] >= vmfDevType(0)) * vmfDevType(1);
        __syncthreads();
    }

    // Tanh is defined in cuda/std/cmath.

    template <typename vmfDevType>
    __global__ void convolute1D(vmfDevType* input, vmfDevType* kernel, vmfDevType* output, cuSize inputSize, cuSize kernelSize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        cuSize outputSize = inputSize - kernelSize + 1;
        if (threadIndexX < outputSize)
        {
            vmfDevType sum(0);
            for (cuSize j = 0; j < kernelSize; ++j) sum += input[threadIndexX + j] * kernel[j];
            output[threadIndexX] = sum;
        }
        __syncthreads();
    }

    template <typename vmfDevType>
    __global__ void dotProduct(vmfDevType* vars0, vmfDevType* vars1, vmfDevType* result, cuSize arraySize)
    {
        cuSize threadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
        *result = 0;
        if (i < arraySize) *result += vars0[i] * vars1[i];
        __syncthreads();
    }
}
